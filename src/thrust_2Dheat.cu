#include "hip/hip_runtime.h"
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/functional.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>

#include <iomanip>
#include <typeinfo>
#include <time.h>

#include <tiled.h>
#include <printmatrix.hpp>
#include <init.cu>

# define L_x 1
# define L_y 1
# define N_x 1024
# define N_y 1024

struct temperature_update_functor{

private:
    double dx, dy, dt, alpha;

public:
    temperature_update_functor(double _dx, double _dy, 
        double _dt, double _alpha) : dx(_dx), dy(_dy), dt(_dt), alpha(_alpha){}
     
    template <typename Tuple>
    __host__ __device__
    void operator() (Tuple t){  
        if(thrust::get<5>(t)){
            thrust::get<0>(t) += 
            (alpha*dt)*
            ((1/dy)*(thrust::get<1>(t)-2*thrust::get<0>(t)+thrust::get<4>(t))+
            (1/dx)*(thrust::get<2>(t)-2*thrust::get<0>(t)+thrust::get<3>(t))); 
        }
    }
};



int main(){

    double  dx = (double)L_x/N_x,
            dy = (double)L_y/N_y,
            alpha = 0.2,
            dt = 1;

    int nsteps = 10000;

    clock_t startclock, stopclock;
    double timeperstep;

    thrust::device_vector<double> A= init_temp(N_x, N_y);

    thrust::device_vector<int> stencil(N_x, 1);
    std::cout<<typeid(stencil).name()<<std::endl;
    stencil[0] = 0; stencil[N_x-1] = 0;

    // Create stencil iterators
    typedef thrust::device_vector<int>::iterator IntIterator;
    typedef thrust::device_vector<double>::iterator DoubleIterator;
    typedef tiled_range<IntIterator> StencilIterator;

    StencilIterator repeated_stencil(stencil.begin(), stencil.end(), N_x-1);

    startclock = clock();

    // Update temperatures 
    for(int t=0; t<nsteps; t+=dt){
        DoubleIterator s1 = A.begin()+N_y;
        DoubleIterator s2 = A.begin()+N_y - N_y;
        DoubleIterator s3 = A.begin()+N_y - 1;
        DoubleIterator s4 = A.begin()+N_y + 1;
        DoubleIterator s5 = A.begin()+N_y + N_y;

        typedef thrust::tuple<DoubleIterator, DoubleIterator, DoubleIterator, 
                              DoubleIterator, DoubleIterator, 
                              StencilIterator::iterator > IteratorTuple;
        
        thrust::zip_iterator<IteratorTuple> zip = 
        make_zip_iterator(thrust::make_tuple(s1, s2, s3, s4, s5, 
                                                repeated_stencil.begin()));

        thrust::for_each(zip, zip+N_y*N_x-2*N_x, 
                        temperature_update_functor(dx, dy, dt, alpha));
        
    }

    stopclock = clock();
    timeperstep =((double)(stopclock-startclock))/CLOCKS_PER_SEC;
    timeperstep = timeperstep / nsteps;
    timeperstep = timeperstep / (N_x*N_y);

    printf("Time per point per step = %e\n",timeperstep);


    thrust::host_vector<double> A_h = A;
    write_to_file(A_h.data(), N_y, N_x);

    return 0;
}


