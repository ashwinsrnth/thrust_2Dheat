#include "hip/hip_runtime.h"


/**
    thrust_2Dheat.cu
    ----------------

    Thrust based GPU solver for the 2-D heat equation, solved
    by explicit finite differences.

    This program runs the simulation. The 
    temperature_update_functor is defined here.
**/

#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/functional.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>

#include <stdlib.h>
#include <iomanip>
#include <typeinfo>
#include <time.h>
#include <stdio.h>

#include <tiled.h>
#include <init.h>
#include <functors.h>
#include <printmatrix.hpp>

int main(int argc, char* argv[]){

    double  L_x = 1024,
            L_y = 1024,
            N_x = 1024,
            N_y = 1024;

    // Read N_x, N_y from command line. If none provided
    // then taken from definition above.

    if (argc == 3){             // Override definitions for N_x, N_y
        N_x = atof(argv[1]);
        N_y = atof(argv[2]);
        L_x = N_x;
        L_y = N_y;
    }

    double  dx = (double)L_x/N_x,
            dy = (double)L_y/N_y,
            alpha = 0.02,
            dt = 1;

    int nsteps = 100;

    clock_t startclock, stopclock;
    double timeperstep;

    // Initialise temperatures in host memory (init.cu)
    thrust::host_vector<double> A_h(N_x*N_y);
    init_temp(thrust::raw_pointer_cast(A_h.data()), N_x, N_y);
    thrust::device_vector<double> A(A_h.begin(), A_h.end());

    // Create iterator for stencil which describes the boundaries (see README)
    thrust::device_vector<int> stencil(N_x, 1);
    stencil[0] = 0; stencil[N_x-1] = 0;

    typedef thrust::device_vector<int>::iterator IntIterator;
    typedef thrust::device_vector<double>::iterator DoubleIterator;
    typedef tiled_range<IntIterator> StencilIterator;
    typedef thrust::tuple<DoubleIterator, DoubleIterator, DoubleIterator, 
                          DoubleIterator, DoubleIterator, 
                          StencilIterator::iterator > IteratorTuple;
    
    typedef thrust::tuple<DoubleIterator, DoubleIterator> DoubleTuple;
    typedef thrust::zip_iterator<DoubleTuple> DoubleZipIterator;
    typedef thrust::tuple<DoubleIterator, DoubleZipIterator, DoubleZipIterator,
                          StencilIterator::iterator> StencilTuple;

    StencilIterator repeated_stencil(stencil.begin(), stencil.end(), N_y-1);

    // Temperature update loop:
    startclock = clock();
    
    for(int t=0; t<nsteps; t+=dt){
        DoubleIterator s1 = A.begin()+N_x;
        DoubleIterator s2 = A.begin()+N_x - N_x;
        DoubleIterator s3 = A.begin()+N_x - 1;
        DoubleIterator s4 = A.begin()+N_x + 1;
        DoubleIterator s5 = A.begin()+N_x + N_x;

        thrust::zip_iterator<StencilTuple> zip =
        make_zip_iterator(thrust::make_tuple(s1, 
                          thrust::make_zip_iterator(make_tuple(s3, s4)),
                          thrust::make_zip_iterator(make_tuple(s2, s5)),
                          repeated_stencil.begin()));

        thrust::for_each(zip, zip+N_y*N_x-2*N_x, 
                        temperature_update_functor2(dx, dy, dt, alpha));
        
    }
    stopclock = clock();
    
    // Calculate time per point per step:
    timeperstep =((double)(stopclock-startclock))/CLOCKS_PER_SEC;
    timeperstep = timeperstep / nsteps;
    timeperstep = timeperstep / (N_x*N_y);

    printf("Time per point per step = %e\n",timeperstep);
    
    // Copy results to host and write to file:
    A_h = A;
    write_to_file(A_h.data(), N_y, N_x);
    return 0;
}


