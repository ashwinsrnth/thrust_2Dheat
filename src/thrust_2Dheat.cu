#include "hip/hip_runtime.h"
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/functional.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>

#include <iomanip>
#include <typeinfo>
#include <time.h>

#include <tiled.h>
#include <printmatrix.hpp>
#include <init.cu>

# define L_x 1
# define L_y 1
# define N_x 1024
# define N_y 1024

struct temperature_update_functor{

private:
    double dx, dy, dt, alpha;

public:
    temperature_update_functor(double _dx, double _dy, 
        double _dt, double _alpha) : dx(_dx), dy(_dy), dt(_dt), alpha(_alpha){}
     
    template <typename Tuple>
    __host__ __device__
    void operator() (Tuple t){  
        if(thrust::get<5>(t)){
            thrust::get<0>(t) += 
            (alpha*dt)*
            ((1/dy)*(thrust::get<1>(t)-2*thrust::get<0>(t)+thrust::get<4>(t))+
            (1/dx)*(thrust::get<2>(t)-2*thrust::get<0>(t)+thrust::get<3>(t))); 
        }
    }
};


int main(){

    double  dx = (double)L_x/N_x,
            dy = (double)L_y/N_y,
            alpha = 0.2,
            dt = 1;

    int nsteps = 10000;

    clock_t startclock, stopclock;
    double timeperstep;

    // Initialise temperatures in device memory (init.cu)
    thrust::device_vector<double> A = init_temp(N_x, N_y);

    // Create iterator for stencil which describes the boundaries (see README)
    thrust::device_vector<int> stencil(N_x, 1);
    stencil[0] = 0; stencil[N_x-1] = 0;

    typedef thrust::device_vector<int>::iterator IntIterator;
    typedef thrust::device_vector<double>::iterator DoubleIterator;
    typedef tiled_range<IntIterator> StencilIterator;

    StencilIterator repeated_stencil(stencil.begin(), stencil.end(), N_x-1);

    // Temperature update loop:
    startclock = clock();
    
    for(int t=0; t<nsteps; t+=dt){
        DoubleIterator s1 = A.begin()+N_y;
        DoubleIterator s2 = A.begin()+N_y - N_y;
        DoubleIterator s3 = A.begin()+N_y - 1;
        DoubleIterator s4 = A.begin()+N_y + 1;
        DoubleIterator s5 = A.begin()+N_y + N_y;

        typedef thrust::tuple<DoubleIterator, DoubleIterator, DoubleIterator, 
                              DoubleIterator, DoubleIterator, 
                              StencilIterator::iterator > IteratorTuple;
        
        thrust::zip_iterator<IteratorTuple> zip = 
        make_zip_iterator(thrust::make_tuple(s1, s2, s3, s4, s5, 
                                                repeated_stencil.begin()));

        thrust::for_each(zip, zip+N_y*N_x-2*N_x, 
                        temperature_update_functor(dx, dy, dt, alpha));
        
    }
    stopclock = clock();
    
    // Calculate time per point per step:
    timeperstep =((double)(stopclock-startclock))/CLOCKS_PER_SEC;
    timeperstep = timeperstep / nsteps;
    timeperstep = timeperstep / (N_x*N_y);

    printf("Time per point per step = %e\n",timeperstep);

    // Copy results to host and write to file:
    thrust::host_vector<double> A_h = A;
    write_to_file(A_h.data(), N_y, N_x);

    return 0;
}


