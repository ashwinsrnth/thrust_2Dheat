#include "hip/hip_runtime.h"


/**
    thrust_2Dheat.cu
    ----------------

    Thrust based GPU solver for the 2-D heat equation, solved
    by explicit finite differences.

    This program runs the simulation. The 
    temperature_update_functor is defined here.
**/

#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/functional.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>

#include <stdlib.h>
#include <iomanip>
#include <typeinfo>
#include <time.h>

#include <tiled.h>
#include <printmatrix.hpp>
#include <init.cu>


struct temperature_update_functor{

private:
    double dx, dy, dt, alpha;

public:
    temperature_update_functor(double _dx, double _dy, 
        double _dt, double _alpha) : dx(_dx), dy(_dy), dt(_dt), alpha(_alpha){}
     
    template <typename Tuple>
    __host__ __device__
    void operator() (Tuple t){  
        if(thrust::get<5>(t)){
            thrust::get<0>(t) += 
            (alpha*dt)*
            ((1/(dy*dy))*(thrust::get<1>(t)-2*thrust::get<0>(t)+thrust::get<4>(t))+
            (1/(dx*dx))*(thrust::get<2>(t)-2*thrust::get<0>(t)+thrust::get<3>(t))); 
        }
    }
};


int main(int argc, char* argv[]){

    double  L_x = 1024,
            L_y = 1024,
            N_x = 1024,
            N_y = 1024;

    // Read N_x, N_y from command line. If none provided
    // then taken from definition above.

    if (argc == 3){             // Override definitions for N_x, N_y
        N_x = atof(argv[1]);
        N_y = atof(argv[2]);
        L_x = N_x;
        L_y = N_y;
    }

    double  dx = (double)L_x/N_x,
            dy = (double)L_y/N_y,
            alpha = 0.2,
            dt = 1;

    int nsteps = 10000;

    clock_t startclock, stopclock;
    double timeperstep;

    // Initialise temperatures in device memory (init.cu)
    thrust::device_vector<double> A = init_temp(N_x, N_y);

    // Create iterator for stencil which describes the boundaries (see README)
    thrust::device_vector<int> stencil(N_x, 1);
    stencil[0] = 0; stencil[N_x-1] = 0;

    typedef thrust::device_vector<int>::iterator IntIterator;
    typedef thrust::device_vector<double>::iterator DoubleIterator;
    typedef tiled_range<IntIterator> StencilIterator;

    StencilIterator repeated_stencil(stencil.begin(), stencil.end(), N_y-1);

    // Temperature update loop:
    startclock = clock();
    
    for(int t=0; t<nsteps; t+=dt){
        DoubleIterator s1 = A.begin()+N_x;
        DoubleIterator s2 = A.begin()+N_x - N_x;
        DoubleIterator s3 = A.begin()+N_x - 1;
        DoubleIterator s4 = A.begin()+N_x + 1;
        DoubleIterator s5 = A.begin()+N_x + N_x;

        typedef thrust::tuple<DoubleIterator, DoubleIterator, DoubleIterator, 
                              DoubleIterator, DoubleIterator, 
                              StencilIterator::iterator > IteratorTuple;
        
        thrust::zip_iterator<IteratorTuple> zip = 
        make_zip_iterator(thrust::make_tuple(s1, s2, s3, s4, s5, 
                                                repeated_stencil.begin()));

        thrust::for_each(zip, zip+N_y*N_x-2*N_x, 
                        temperature_update_functor(dx, dy, dt, alpha));
        
    }
    stopclock = clock();
    
    // Calculate time per point per step:
    timeperstep =((double)(stopclock-startclock))/CLOCKS_PER_SEC;
    timeperstep = timeperstep / nsteps;
    timeperstep = timeperstep / (N_x*N_y);

    printf("Time per point per step = %e\n",timeperstep);

    // Copy results to host and write to file:
    thrust::host_vector<double> A_h = A;
    write_to_file(A_h.data(), N_y, N_x);

    return 0;
}


